#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

int *host_A, *host_C1, *host_C2;       // host data
int *device_A, *device_C;   // results

__global__ void vecAddOne(int *A, int *C, int N)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
    
   if( i < N )
      C[i] = A[i] + 1; 
}

void vecAddOne_h(int *A1, int *C1, int N)
{
   for(int i=0;i<N;i++)
      C1[i] = A1[i] + 1;
}

int main(int argc,char **argv)
{
   int n=1024*1024;
   int nBytes = n*sizeof(int);
   int block_size = 32, block_no = n / block_size; 

   // ===============================================================
   // CPU 메모리 설정 
   //
   host_A = (int *)malloc(nBytes);
   host_C1 = (int *)malloc(nBytes);    
   host_C2 = (int *)malloc(nBytes);    

   // ===============================================================    
   printf("Allocating device memory on host..\n");
   hipMalloc((void **)&device_A, n*sizeof(int));
   hipMalloc((void **)&device_C, n*sizeof(int));
   // ===============================================================    
   printf("Copying to device..\n");
   hipMemcpy(device_A, host_A, n*sizeof(int),hipMemcpyHostToDevice);
   // ===============================================================
   printf("Doing GPU Vector + 1 \n");
   vecAddOne<<<block_no,block_size>>>(device_A, device_C, n);   
   hipDeviceSynchronize();
   // ===============================================================
   printf("Doing a CPU Vector add\n");    
   vecAddOne_h(host_A, host_C1, n);
   
   hipMemcpy(host_C2, device_C, n*sizeof(int), hipMemcpyDeviceToHost);

   // 결과 비교
   printf("결과 비교\n");
   for(int i=0; i<n;i++)
   {
       if(host_C1[i] != host_C2[i])
       {
           printf("Something Wrong ! \n");
           break;
       }
   }
   hipFree(device_A);
   hipFree(device_C);
   free(host_A);
   free(host_C1);
   free(host_C2);
   return 0;
}  
