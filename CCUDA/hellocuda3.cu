
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloCUDA(void)
{
  printf("Hello thread %d in block %d\n", threadIdx.x, blockIdx.x);
}

int main()
{
  int n = 12;
  int blockDim = 4;            // Block내의 Thread의 수
  int gridDim = n / blockDim;  // Grid에서 Block의 수
  
  // 따라서, 전체 생성 thread의 수는 blockDim * threadDim  
    
  helloCUDA<<<gridDim, blockDim>>>();
    
  hipDeviceSynchronize();
  return 0;
}
